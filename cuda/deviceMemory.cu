
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>
constexpr const int N=64;


__global__ void VecAdd(float* A,float* B,float* C)
{
  int i=blockDim.x*blockIdx.x+threadIdx.x;
  if(i<N) C[i]=A[i]+B[i];
}

__global__ void PitchAdd(float* devPtr, size_t pitch,int width, int height)
{
  for(int i=0;i<height;++i)
  {
    auto row=(float*)((char*)devPtr+i*pitch);
    for(int j=0;j<width;++j)
    {
      auto element=sin(row[j]);
    }
  }
}

__global__ void D3Add(hipPitchedPtr devPitchedPtr, int width, int height, int depth)
{
  auto devPtr=(char*)devPitchedPtr.ptr;
  size_t pitch=devPitchedPtr.pitch;
  size_t slicePitch=pitch*height;

  for(int i=0;i<depth;++i)
  {
    char* slice=devPtr+i*slicePitch;
    for(int j=0;j<height;++j)
    {
      auto row=(float*)(slice+j*pitch);
      for(int k=0;k<width;++k)
      {
	auto element=row[k];
      }
    }
  }
}

int main()
{
  size_t  size= sizeof(float)*N; 
  auto A= new float[N];
  auto B= new float[N];
  auto C= new float[N];
 
  for(int i=0;i<N;++i)
  {
    A[i]=B[i]=i;
  }
  float* d_A,*d_B,*d_C;
  hipMalloc(&d_A,size);
  hipMalloc(&d_B,size);
  hipMalloc(&d_C,size);
  hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
  hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);

  int threadPerBlock=64;
  int blockPerGrid=(N+threadPerBlock-1) / threadPerBlock ;

  VecAdd<<<blockPerGrid,threadPerBlock>>>(d_A,d_B,d_C);
  hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  int width=64,height=64;
  float* devPtr;
  size_t pitch;
  hipMallocPitch(&devPtr,&pitch,width*sizeof(float),height);
  printf("pitch=%d\n",pitch);
  PitchAdd<<<100,512>>>(devPtr,pitch,width,height);

  hipFree(devPtr);

  

  int depth=64;
  hipExtent extent=make_hipExtent(width*sizeof(float),height,depth);
  hipPitchedPtr devPitchedPtr;
  hipMalloc3D(&devPitchedPtr,extent);
  D3Add<<<100,512>>>(devPitchedPtr,width,height,depth);

  return 0;
}
